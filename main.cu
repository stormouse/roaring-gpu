
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdint>

__global__ void mallocTest()
{
    size_t size = 123;
    char* ptr = (char*)malloc(size);
    memset(ptr, 0, size);
    printf("Thread %d got pointer: %p\n", threadIdx.x, ptr);
    free(ptr);
}

int main()
{
    // Set a heap size of 128 megabytes. Note that this must
    // be done before any kernel is launched.
    hipDeviceSetLimit(hipLimitMallocHeapSize, 128*1024*1024);
    mallocTest<<<1, 5>>>();
    hipDeviceSynchronize();
    return 0;
}