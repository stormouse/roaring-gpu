#include "bitmap_example.cuh"

int main()
{
    hipDeviceSetLimit(hipLimitMallocHeapSize, (1024 * 1024 * 1024UL));

    testBitmapIntersect();

    hipDeviceSynchronize();

    testBitmapUnion();

    return 0;
}