#include "bitmap_example.cuh"
#include <iostream>

int main()
{
    hipDeviceSetLimit(hipLimitMallocHeapSize, (2048 * 1024 * 1024UL));

    // malloc_test1();
    // hipDeviceSynchronize();
    // malloc_test2();
    // hipDeviceSynchronize();

    testBitmapUnion();

    hipDeviceSynchronize();

    std::cout << "\n----------------dividing-line----------------\n" << "\n";

    testBitmapIntersect();
    
    hipDeviceSynchronize();

    return 0;
}