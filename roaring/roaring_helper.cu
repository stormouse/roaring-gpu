#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <random>
#include <vector>
#include "bitop.cuh"
#include "cuda_common.cuh"
#include "memory.cuh"
#include "roaring_helper.cuh"

namespace tora::roaring
{

__global__ void initializeIntermediateResultContainers(RoaringBitmapFlat* bitmap, int containerLow, int containerHigh)
{
    int n = containerHigh - containerLow;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int step = gridDim.x * blockDim.x;
    while (idx < n)
    {
        bitmap->containers[containerLow + idx].data = (uint32_t*)custom_malloc(65536);
        bitmap->containers[containerLow + idx].capacity = 8192;
        bitmap->containers[containerLow + idx].cardinality = 0;
        idx += step;
    }
}

__global__ void buildRandomArrayContainers(
    RoaringBitmapFlat* bitmap, int* containerIndexes, int n, int arrayElementLow, int arrayElementHigh)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    while (idx < n)
    {
        hiprandState state;
        hiprand_init(42, idx, 0, &state);
        float randomValue = hiprand_uniform(&state);

        int numElements = arrayElementLow + static_cast<int>((arrayElementHigh - arrayElementLow) * randomValue);

        Container& dst = bitmap->containers[containerIndexes[idx]];
        dst.capacity = (numElements + 1) / 2;
        dst.data = (uint32_t*)custom_malloc(dst.capacity * sizeof(uint32_t));
        dst.type = ContainerType::Array;
        dst.cardinality = 0;
        
        for (int i = 0; i < dst.capacity; i++)
        {
            dst.data[i] = 0;
        }

        for (int i = 0; i < numElements; i++)
        {
            array_setBit(dst, i, true);
        }

        idx += gridDim.x * blockDim.x;
    }
}

__global__ void buildRandomBitsetContainers(RoaringBitmapFlat* bitmap, int* containerIndexes, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    while (idx < n)
    {
        hiprandState state;
        hiprand_init(42, idx, 0, &state);

        Container& dst = bitmap->containers[containerIndexes[idx]];
        dst.capacity = 2048;
        dst.data = (uint32_t*)custom_malloc(dst.capacity * sizeof(uint32_t));
        dst.type = ContainerType::Bitset;
        dst.cardinality = 0;

        for (int i = 0; i < dst.capacity; i++)
        {
            uint32_t value = (uint32_t)(hiprand_uniform(&state) * (uint64_t)(UINT_MAX));
            dst.data[i] = value;
            dst.cardinality += bitsSet(value);
        }

        idx += gridDim.x * blockDim.x;
    }
}

RoaringBitmapDevice getRandomRoaringBitmap(
    int containerLow, int containerHigh, int numArrays, int numBitsets, int arrayElementLow, int arrayElementHigh)
{
    static const int ARRAY = 0;
    static const int BITSET = 2;

    std::random_device rd;
    std::mt19937 g(rd());

    std::vector<int> containerIndex(containerHigh - containerLow);
    std::vector<int> containerType(numArrays + numBitsets);

    for (int i = 0; i < containerHigh - containerLow; i++)
    {
        containerIndex[i] = containerLow + i;
    }

    for (int i = 0; i < numArrays; i++)
    {
        containerType[i] = ARRAY;
    }

    for (int i = numArrays; i < numArrays + numBitsets; i++)
    {
        containerType[i] = BITSET;
    }

    std::shuffle(containerIndex.begin(), containerIndex.end(), g);
    std::shuffle(containerType.begin(), containerType.end(), g);

    int numContainers = std::min(containerHigh - containerLow, numArrays + numBitsets);

    std::vector<int> arrayIndexes;
    std::vector<int> bitsetIndexes;

    for (int i = 0; i < numContainers; i++)
    {
        if (containerType[i] == ARRAY)
        {
            arrayIndexes.push_back(containerIndex[i]);
        }
        else
        {
            bitsetIndexes.push_back(containerIndex[i]);
        }
    }

    int* arrayIndexes_d;
    int* bitsetIndexes_d;
    checkCuda(hipMalloc((void**)&arrayIndexes_d, sizeof(int) * numArrays));
    checkCuda(hipMalloc((void**)&bitsetIndexes_d, sizeof(int) * numBitsets));
    checkCuda(
        hipMemcpy(arrayIndexes_d, arrayIndexes.data(), sizeof(int) * arrayIndexes.size(), hipMemcpyHostToDevice));
    checkCuda(
        hipMemcpy(bitsetIndexes_d, bitsetIndexes.data(), sizeof(int) * bitsetIndexes.size(), hipMemcpyHostToDevice));

    RoaringBitmapDevice bitmap;

    int threadsPerBlock = 256;
    int blocksPerGrid = 64;

    buildRandomArrayContainers<<<blocksPerGrid, threadsPerBlock>>>(
        bitmap.devPtr(), arrayIndexes_d, arrayIndexes.size(), arrayElementLow, arrayElementHigh);
    buildRandomBitsetContainers<<<blocksPerGrid, threadsPerBlock>>>(
        bitmap.devPtr(), bitsetIndexes_d, bitsetIndexes.size());

    checkCuda(hipFree(arrayIndexes_d));
    checkCuda(hipFree(bitsetIndexes_d));

    hipDeviceSynchronize();

    return bitmap;
}


RoaringBitmapDevice getIntermediateBitmap(int containerLow, int containerHigh)
{
    RoaringBitmapDevice bitmap;
    initializeIntermediateResultContainers<<<256, 64>>>(bitmap.devPtr(), containerLow, containerHigh);
    return bitmap;
}

}  // namespace tora::roaring